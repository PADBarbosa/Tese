extern "C" { 


#include <hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>

	__global__ void multiply(float* input, float* output, int* sizebuffer) {  //é necessária distinção do M e N porque pode nao ser quadrado
		int size = sizebuffer[0];

		for(int i = 0; i < size; i++){
			output[i] = input[i]*3;
		}
	}

	__global__ void multiply2(float* input, float* output, int* sizebuffer) {  //é necessária distinção do M e N porque pode nao ser quadrado
		int size = sizebuffer[0];

		for(int i = 0; i < size; i++){
			output[i] = input[i]*4;
		}
	}
}
