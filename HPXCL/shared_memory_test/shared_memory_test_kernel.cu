#include "hip/hip_runtime.h"
extern "C" { 
	__global__ void dynamicReverse(int* d, int* sizebuffer){
		int n = sizebuffer[0];

		extern __shared__ int s[64];

		int t = threadIdx.x;
		int tr = n-t-1;
		
		//d[t] = d[tr];
		
		s[t] = d[t];
		
		__syncthreads();

		d[t] = s[tr];
	}
}