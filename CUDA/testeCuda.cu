
#include <hip/hip_runtime.h>
#include <iostream>

//Usado para testar se o sistema consegue executar código CUDA sem problemas


__global__ void add( int a, int b, int* c ) {
	*c = a + b;
}

int main( void ) {
	int* c;
    hipHostMalloc((void**)&c, sizeof(int) * 1, hipHostMallocDefault);


	int* dev_c;
	hipMalloc( (void**)&dev_c, sizeof(int) );

	add<<<1,1>>>( 2, 11, dev_c );

	hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost );

	printf( "2 + 11 = %d\n", *c );

	hipFree( dev_c );

	return 0;
}