
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

void cudaCheckError() {
	hipError_t e=hipGetLastError();
	if(e!=hipSuccess) {
		printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));
		exit(0);
	}
}

__global__ void multiply(float* input, float* output, int size) {
	for(int i = 0; i < size/blockDim.x; i++){
		output[threadIdx.x * (size/blockDim.x) + i] = input[threadIdx.x * (size/blockDim.x) + i]*2;
		//output[i] = input[i]*2;
	}
}

__global__ void multiply2(float* input, float* output, int size) {
	for(int i = 0; i < size/blockDim.x; i++){
		output[threadIdx.x * (size/blockDim.x) + i] = input[threadIdx.x * (size/blockDim.x) + i]*4;
	}
}


int main( void ) {
	int size = 8;

	auto start = std::chrono::steady_clock::now();
	
	float* a;
	a = (float*) malloc(sizeof(float)*size);
	cudaCheckError();

	for(int i = 0; i < size; i++){
		a[i] = 1;
		std::cout << a[i] << ", ";
	}
	std::cout << std::endl;

	float* b;
	b = (float*) malloc(sizeof(float)*size);
	cudaCheckError();

	float* c;
	c = (float*) malloc(sizeof(float)*size);
	cudaCheckError();



	float* dev_a;
	hipMalloc( (void**)&dev_a, sizeof(float)*size );
	cudaCheckError();

	float* dev_b;
	hipMalloc( (void**)&dev_b, sizeof(float)*size );
	cudaCheckError();

	float* dev_c;
	hipMalloc( (void**)&dev_c, sizeof(float)*size );
	cudaCheckError();



	//destino source, size

	hipMemcpy( dev_a, a, sizeof(float)*size, hipMemcpyHostToDevice );
	cudaCheckError();

	hipMemcpy( dev_b, a, sizeof(float)*size, hipMemcpyHostToDevice );
	cudaCheckError();

	multiply<<<1,1>>>( dev_a, dev_b, size );
	cudaCheckError();


	hipMemcpy( b, dev_b, sizeof(float)*size, hipMemcpyDeviceToHost );
	cudaCheckError();

	for(int i = 0; i < size; i++){
		std::cout << b[i] << ", ";
	}
	std::cout << std::endl;





	hipMemcpy( dev_a, a, sizeof(float)*size, hipMemcpyHostToDevice );
	cudaCheckError();
	
	multiply2<<<1,1>>>( dev_a, dev_c, size );
	
	hipMemcpy( c, dev_c, sizeof(float)*size, hipMemcpyDeviceToHost );
	cudaCheckError();

	for(int i = 0; i < size; i++){
		std::cout << c[i] << ", ";
	}
	std::cout << std::endl;

	auto end = std::chrono::steady_clock::now();
	std::chrono::duration<double> elapsed_seconds = end-start;
	std::cout << "elapsed time: " << elapsed_seconds.count() << "s\n";

	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );
	return 0;
}